#include "hip/hip_runtime.h"
__global__ void Compute_eigsts_hom(hipfftDoubleComplex *eigsts00, 
                                   hipfftDoubleComplex *eigsts10, 
                                   hipfftDoubleComplex *eigsts20, 
                                   hipfftDoubleComplex *dfdphi_d, 
                                   double Chom11_d, double Chom12_d, 
                                   double Chom44_d, double epszero_d, 
                                   int ny_d, int nz_d)
{

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (nz_d)*(j + i*(ny_d));

  double eig11, eig22, eig33, hphi, e_temp;
  
  e_temp = dfdphi_d[idx].x;

  hphi = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  eig11 = epszero_d * hphi; 
  eig22 = epszero_d * hphi; 
  eig33 = epszero_d * hphi; 

  eigsts00[idx].x = (Chom11_d)*eig11 + 
                    (Chom12_d)*eig22 +
                    (Chom12_d)*eig33 ;     

  eigsts10[idx].x = (Chom12_d)*eig11 + 
                    (Chom11_d)*eig22 +
                    (Chom12_d)*eig33 ;     

  eigsts20[idx].x = (Chom12_d)*eig11 + 
                    (Chom12_d)*eig22 +
                    (Chom11_d)*eig33 ;

  eigsts00[idx].y = 0.0;    
  eigsts10[idx].y = 0.0;    
  eigsts20[idx].y = 0.0;    

}

__global__ void Initialize_disp(hipfftDoubleComplex *ux_d, 
                                hipfftDoubleComplex *uy_d,
                                hipfftDoubleComplex *uz_d, int ny_d, int nz_d)
{

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (nz_d)*(j + i*(ny_d));

  ux_d[idx].x = 0.0;
  uy_d[idx].x = 0.0;
  uz_d[idx].x = 0.0;

  ux_d[idx].y = 0.0;
  uy_d[idx].y = 0.0;
  uz_d[idx].y = 0.0;

}

__global__ void Compute_uzero(int ny_d, int nz_d, 
                          hipfftDoubleComplex *ux_d, hipfftDoubleComplex *uy_d,
                          hipfftDoubleComplex *uz_d, double *kx_d, double *ky_d,
                          double *kz_d, hipfftDoubleComplex *eigsts00, 
                          hipfftDoubleComplex *eigsts10, 
                          hipfftDoubleComplex *eigsts20,
                          double Chom11_d, double Chom12_d, double Chom44_d)
{

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (nz_d)*(j + i*(ny_d));

  double               adjomega[6], det_omega, invomega_v[6];
  double               nk[3];
  double               omega[6];
  hipfftDoubleComplex   eig_v[3], fk10, fk20, fk30;
  
  nk[0] = (double)kx_d[i];
  nk[1] = (double)ky_d[j];
  nk[2] = (double)kz_d[k];

  invomega_v[0] = (Chom11_d)*nk[0]*nk[0] + (Chom44_d)*nk[1]*nk[1] +
                  (Chom44_d)*nk[2]*nk[2];
  invomega_v[1] = (Chom44_d)*nk[0]*nk[0] + (Chom11_d)*nk[1]*nk[1] +
                  (Chom44_d)*nk[2]*nk[2];
  invomega_v[2] = (Chom44_d)*nk[0]*nk[0] + (Chom44_d)*nk[1]*nk[1] +
                  (Chom11_d)*nk[2]*nk[2];
  invomega_v[3] = ((Chom12_d) + (Chom44_d))*nk[1]*nk[2];
  invomega_v[4] = ((Chom12_d) + (Chom44_d))*nk[0]*nk[2];
  invomega_v[5] = ((Chom12_d) + (Chom44_d))*nk[0]*nk[1];

  det_omega = invomega_v[0]*(invomega_v[1]*invomega_v[2] -
                             invomega_v[3]*invomega_v[3])-
              invomega_v[5]*(invomega_v[5]*invomega_v[2] -
                             invomega_v[4]*invomega_v[3])+
              invomega_v[4]*(invomega_v[5]*invomega_v[3] -
                             invomega_v[4]*invomega_v[1]);

  adjomega[0] = (invomega_v[1]*invomega_v[2]-
                 invomega_v[3]*invomega_v[3]);
  adjomega[1] = (invomega_v[0]*invomega_v[2]-
                 invomega_v[4]*invomega_v[4]);
  adjomega[2] = (invomega_v[0]*invomega_v[1]-
                 invomega_v[5]*invomega_v[5]);
  adjomega[3] =-(invomega_v[0]*invomega_v[3]-
                 invomega_v[4]*invomega_v[5]);
  adjomega[4] = (invomega_v[5]*invomega_v[3]-
                 invomega_v[4]*invomega_v[1]);
  adjomega[5] =-(invomega_v[5]*invomega_v[2]-
                 invomega_v[4]*invomega_v[3]);

  if (fabs(det_omega) > 1.0e-06){
     omega[0] = (1.0/det_omega)*adjomega[0];
     omega[1] = (1.0/det_omega)*adjomega[1];
     omega[2] = (1.0/det_omega)*adjomega[2];
     omega[3] = (1.0/det_omega)*adjomega[3];
     omega[4] = (1.0/det_omega)*adjomega[4];
     omega[5] = (1.0/det_omega)*adjomega[5];
  }

  else{
     omega[0] = 0.0;
     omega[1] = 0.0;
     omega[2] = 0.0;
     omega[3] = 0.0;
     omega[4] = 0.0;
     omega[5] = 0.0;
  }

  eig_v[0].x = eigsts00[idx].x; 
  eig_v[1].x = eigsts10[idx].x; 
  eig_v[2].x = eigsts20[idx].x; 

  eig_v[0].y = eigsts00[idx].y; 
  eig_v[1].y = eigsts10[idx].y; 
  eig_v[2].y = eigsts20[idx].y; 
 
  fk10.x = eig_v[0].x*nk[0];
  fk20.x = eig_v[1].x*nk[1];
  fk30.x = eig_v[2].x*nk[2];

  fk10.y = eig_v[0].y*nk[0];
  fk20.y = eig_v[1].y*nk[1];
  fk30.y = eig_v[2].y*nk[2];

  ux_d[idx].x = omega[0]*fk10.y +
                omega[5]*fk20.y +
                omega[4]*fk30.y ;  
 
  uy_d[idx].x = omega[5]*fk10.y +
                omega[1]*fk20.y +
                omega[3]*fk30.y ;   

  uz_d[idx].x = omega[4]*fk10.y +
                omega[3]*fk20.y +
                omega[2]*fk30.y ;   

  ux_d[idx].y = -1.0*(omega[0]*fk10.x +
                      omega[5]*fk20.x +
                      omega[4]*fk30.x);   

  uy_d[idx].y = -1.0*(omega[5]*fk10.x +
                      omega[1]*fk20.x +
                      omega[3]*fk30.x);   
 
  uz_d[idx].y = -1.0*(omega[4]*fk10.x +
                      omega[3]*fk20.x +
                      omega[2]*fk30.x);   

}

void Calc_uzero(void){

   int complex_size;

   complex_size = sizeof(hipfftDoubleComplex)*nx*ny*nz;

   hipMalloc((void**)&eigsts00, complex_size);
   hipMalloc((void**)&eigsts10, complex_size);
   hipMalloc((void**)&eigsts20, complex_size);
   
   Compute_eigsts_hom<<< Gridsize,Blocksize >>>(eigsts00, eigsts10, eigsts20, 
                                                dfdphi_d, 
                                                Chom11, Chom12, Chom44, 
                                                epszero, ny, nz);

 /************************************************************
  *          Take eigenstress component to fourier space     * 
  ************************************************************/

   hipfftExecZ2Z(plan, eigsts00, eigsts00, HIPFFT_FORWARD);
   hipfftExecZ2Z(plan, eigsts10, eigsts10, HIPFFT_FORWARD);
   hipfftExecZ2Z(plan, eigsts20, eigsts20, HIPFFT_FORWARD);
 
/************************************************************
*                Initializing displacments                  *
*************************************************************/
   Initialize_disp<<< Gridsize, Blocksize>>>(ux_d, uy_d, uz_d, ny, nz);
/**********************************************************
 *                 Zeroth order displacement              *
 **********************************************************/ 
   Compute_uzero<<< Gridsize, Blocksize >>>(ny, nz, ux_d, uy_d, uz_d, kx_d, 
                          ky_d, kz_d, eigsts00, eigsts10, eigsts20,
                          Chom11, Chom12, Chom44);

   hipFree(eigsts00);
   hipFree(eigsts10);
   hipFree(eigsts20);

}
