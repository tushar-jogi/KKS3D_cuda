#include"../include/binary.h"

int main (int argc, char*argv[])
{   
    //Function declarations
    void Get_Input_Parameters (char *fnin, char *fnout);
    void Init_Conf ();
    void Evolve ();
    void Usage(void);

    size_t  complex_size, double_size, float_size, complex_size_elast;
    char finput[15] = "bin1ary";
    char fnin[30]="InputParams", fnout[30]="OutParams";
    FILE *fp;

    if (argc > 1){
    	for (int i=0; i<argc; i++){

        	if (strcmp(argv[i],"-i")==0 && argc == 3){ 
           		strcpy(fnin, argv[i+1]);
           		printf("Reading Input Parameters from %s\n", fnin);
                        break;
        	}
        	else if (strcmp(argv[i],"-i")==0 && argc==2){
           		printf("Input file not provided\n");
           		Usage(); 
                        exit (EXIT_FAILURE);
        	}
        	else if (strcmp(argv[i],"--help")==0){
           		Usage(); 
                        exit (EXIT_FAILURE);
        	}
        
    	}
    }
    else{
      Usage();
      exit (EXIT_FAILURE);
    }

    //if (!(fp = fopen (finput, "r"))) {
    //  printf ("File:%s could not be opened\n", finput);
    //  exit (EXIT_FAILURE);
    //}
    //if(fscanf (fp, "%s", fnin)==1){
    //  printf("Input Parameters Filename:%s\n",fnin);
    //}
    //if(fscanf (fp, "%s", fnout)==1){
    //  printf("Output Parameters Filename:%s\n",fnout);
    //}
    //if (!(fpout = fopen (fnout, "w"))) {
    //  printf ("File:%s could not be opened\n", fnout);
    //  exit (EXIT_FAILURE);
   // }

    //fclose (fp);
    //fclose (fpout);

    //Reading simulation parameters
    Get_Input_Parameters (fnin, fnout);
    
    checkCudaErrors( hipMalloc((void**)&nx_d, sizeof(int)));
    checkCudaErrors( hipMalloc((void**)&ny_d, sizeof(int)));
    checkCudaErrors( hipMalloc((void**)&nz_d, sizeof(int)));
    checkCudaErrors( hipMalloc((void**)&elast_int_d, sizeof(int)));
    checkCudaErrors( hipMalloc((void**)&dt_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&c0_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&w_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&kappa_phi_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&diffusivity_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&relax_coeff_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&ppt_size_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&c_beta_eq_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&c_alpha_eq_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&epszero_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&sizescale_d, sizeof(double)));
    checkCudaErrors( hipMalloc((void**)&Chom11_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&Chom12_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&Chom44_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&Chet11_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&Chet12_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&Chet44_d, sizeof(float)));
    checkCudaErrors( hipMalloc((void**)&sigappl_v_d, 6*sizeof(float)));

    checkCudaErrors(hipMemcpy(nx_d, &nx, sizeof(int),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ny_d, &ny, sizeof(int),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(nz_d, &nz, sizeof(int),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(elast_int_d, &elast_int, sizeof(int),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dt_d, &dt, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c0_d, &c0, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(diffusivity_d, &diffusivity, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(relax_coeff_d, &relax_coeff, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ppt_size_d, &ppt_size, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_beta_eq_d, &c_beta_eq, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_alpha_eq_d, &c_alpha_eq, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(sigappl_v_d, sigappl_v, 6*sizeof(float),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(epszero_d, &epszero, sizeof(float),
          hipMemcpyHostToDevice));

    //sizes of complex and double variables
    complex_size = nx*ny*nz*sizeof(hipDoubleComplex);
    complex_size_elast = nx*ny*nz*sizeof(hipfftComplex);
    double_size  = nx*ny*nz*sizeof(double);
    float_size   = nx*ny*nz*sizeof(float);


    //Allocation of global variables
    comp     = (hipDoubleComplex*) malloc (complex_size);
    dfdphi   = (hipDoubleComplex*) malloc (complex_size); 

    checkCudaErrors(hipMalloc((void**)&comp_d, complex_size));
    checkCudaErrors(hipMalloc((void**)&phi_d, complex_size));
    checkCudaErrors(hipMalloc((void**)&dfdphi_d, complex_size));
    checkCudaErrors(hipMalloc((void**)&dfdc_d, complex_size));

    //Allocation of elasticity varaibles
    if (elast_int == 1){

      checkCudaErrors(hipMalloc((void**)&ux_d, complex_size_elast));
      checkCudaErrors(hipMalloc((void**)&uy_d, complex_size_elast));
      checkCudaErrors(hipMalloc((void**)&uz_d, complex_size_elast));
      checkCudaErrors(hipMalloc((void**)&dfeldphi_d, complex_size_elast));
      //checkCudaErrors(hipMalloc((void**)&omega_v0, float_size));
      //checkCudaErrors(hipMalloc((void**)&omega_v1, float_size));
      //checkCudaErrors(hipMalloc((void**)&omega_v2, float_size));
      //checkCudaErrors(hipMalloc((void**)&omega_v3, float_size));
      //checkCudaErrors(hipMalloc((void**)&omega_v4, float_size));
      //checkCudaErrors(hipMalloc((void**)&omega_v5, float_size));

    }

    //Generating initial profile
    Init_Conf();

    checkCudaErrors(hipMalloc((void**)&kx_d, nx*sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&ky_d, ny*sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&kz_d, nz*sizeof(double)));

    kappa_phi = 3.0/(2.0*alpha) *(interface_energy*Ln);
    printf("Kappa phi = %lf\n", kappa_phi);
    
    w = 6.0 * alpha * interface_energy / Ln;
    printf("Barrier potential = %lf\n", w);
     
    checkCudaErrors(hipMemcpy(w_d,  &w,  sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(kappa_phi_d,  &kappa_phi,  sizeof(double),
          hipMemcpyHostToDevice));

    sim_time = 0.0;

    calc_uzero = 1; 

    //Declaring fft plan
    hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_Z2Z); 
    hipfftPlan3d(&elast_plan, nx, ny, nz, HIPFFT_C2C); 

    hipDeviceSynchronize();
    //call to evolve
    Evolve ();

    free (comp);
    free (dfdphi);
    hipFree (comp_d);
    hipFree (phi_d);
    hipFree (dfdc_d);
    hipFree (kx_d);
    hipFree (ky_d);
    hipFree (kz_d);
    hipFree (dfdphi_d);

    hipFree(nx_d );
    hipFree(ny_d );
    hipFree(nz_d );
    hipFree(dt_d );
    hipFree(c0_d );
    hipFree(w_d );
    hipFree(kappa_phi_d );
    hipFree(ppt_size_d );
    hipFree(c_beta_eq_d );
    hipFree(c_alpha_eq_d );
    hipFree(epszero_d );
    hipFree(Chom11_d );
    hipFree(Chom12_d );
    hipFree(Chom44_d );
    hipFree(Chet11_d );
    hipFree(Chet12_d );
    hipFree(Chet44_d );
    hipFree(sigappl_v_d );

    if (elast_int == 1){

      hipFree (ux_d);
      hipFree (uy_d);
      hipFree (uz_d);
      hipFree (dfeldphi_d);
      //hipFree (omega_v0);
      //hipFree (omega_v1);
      //hipFree (omega_v2);
      //hipFree (omega_v3);
      //hipFree (omega_v4);
      //hipFree (omega_v5);

    }

    return 0;
}

void Usage(void){
     printf("This is the help for KKS3D\n"
            "Options are:\n"
            "    --help: display what you are reading\n"
            "    -i filename: opens the filename for input parameters\n");

}

#include "get_input.cu"
#include "init_conf.cu"
#include "evolve.cu"
