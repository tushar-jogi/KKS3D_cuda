#include "hip/hip_runtime.h"
/**************************************************************************

  Interpolation = 1 : elastic constants as function of composition
  Interpolation = 2 : elastic constants as function of phi 

************************************************************************/
#define TOLERENCE 1.0e-06

__global__ void Compute_Ctotal(hipDoubleComplex *dfdphi_d, int *ny_d, 
                              int *nz_d, double *Ctotal11, double *Ctotal12,
                              double *Ctotal44, double *Chom11_d, 
                              double *Chom12_d, double *Chom44_d,
                              double *Chet11_d, double *Chet12_d,
                              double *Chet44_d)
{

   int i = threadIdx.x + blockDim.x*blockIdx.x;
   int j = threadIdx.y + blockDim.y*blockIdx.y;
   int k = threadIdx.z + blockDim.z*blockIdx.z;

   int idx = k + (*nz_d)*(j + i*(*ny_d));

   double hphi, e_temp;

   e_temp = Re(dfdphi_d[idx]);
   hphi = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp  + 10.0);

   Ctotal11[idx]  =  (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
   Ctotal12[idx]  =  (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
   Ctotal44[idx]  =  (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));
   
}


__global__ void Compute_Sij(double *Cavg11, double *Cavg12, double *Cavg44,
                            double *S11_d, double *S12_d, double *S44_d)
{
  *S11_d = ((*Cavg11) + (*Cavg12))/((*Cavg11)*(*Cavg11) + (*Cavg11)*(*Cavg12)-
           2.0*(*Cavg12)*(*Cavg12));
  *S12_d = (-1.0*(*Cavg12))/((*Cavg11)*(*Cavg11) + (*Cavg11)*(*Cavg12) -
           2.0*(*Cavg12)*(*Cavg12));
  *S44_d = 1.0/(*Cavg44);

}

/*__global__ void Compute_Cinhom(double *Cinhom11, double *Cinhom12, 
                               double *Cinhom44, double *Ctotal11, 
                               double *Ctotal12, double *Ctotal44,
                               double *Chom11_d, double *Chom12_d,
                               double *Chom44_d, int *ny_d, int *nz_d)
{

   int i = threadIdx.x + blockDim.x*blockIdx.x;
   int j = threadIdx.y + blockDim.y*blockIdx.y;
   int k = threadIdx.z + blockDim.z*blockIdx.z;

   int idx = k + (*nz_d)*(j + i*(*ny_d));
    
 
   Cinhom11[idx] = Ctotal11[idx] - (*Chom11_d); 
   Cinhom12[idx] = Ctotal12[idx] - (*Chom12_d); 
   Cinhom44[idx] = Ctotal44[idx] - (*Chom44_d); 
}*/

/*__global__ void Compute_eigstr(hipDoubleComplex *dfdphi_d, double *eigstr0,
                               double *eigstr1, double *eigstr2, 
                               double *epszero_d)
{
   int idx = (threadIdx.x + threadIdx.y * blockDim.x) +
            (blockIdx.x*blockDim.x*blockDim.y);

   double hphi, e_temp;

   e_temp = dfdphi_d[idx].x;

   hphi = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

   eigstr0[idx] = (*epszero_d)*hphi;
   eigstr1[idx] = (*epszero_d)*hphi;
   eigstr2[idx] = (*epszero_d)*hphi;

}*/

__global__ void Compute_perstr(int *ny_d, int *nz_d, double *kx_d, 
                               double *ky_d, double *kz_d, 
                         hipDoubleComplex *unewx_d, hipDoubleComplex *unewy_d,
                         hipDoubleComplex *unewz_d, hipDoubleComplex *str_v0_d,
                         hipDoubleComplex *str_v1_d, hipDoubleComplex *str_v2_d,
                         hipDoubleComplex *str_v3_d, hipDoubleComplex *str_v4_d,
                         hipDoubleComplex *str_v5_d)
{
  	

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double  nk[3];

  nk[0] = kx_d[i];
  nk[1] = ky_d[j];
  nk[2] = kz_d[k];

  str_v0_d[idx].x = -1.0*unewx_d[idx].y*nk[0];
  str_v1_d[idx].x = -1.0*unewy_d[idx].y*nk[1];
  str_v2_d[idx].x = -1.0*unewz_d[idx].y*nk[2];
  str_v3_d[idx].x = -1.0*(unewy_d[idx].y*nk[2] + unewz_d[idx].y*nk[1]);
  str_v4_d[idx].x = -1.0*(unewx_d[idx].y*nk[2] + unewz_d[idx].y*nk[0]);
  str_v5_d[idx].x = -1.0*(unewx_d[idx].y*nk[1] + unewy_d[idx].y*nk[0]);

  str_v0_d[idx].y =  unewx_d[idx].x*nk[0];
  str_v1_d[idx].y =  unewy_d[idx].x*nk[1];
  str_v2_d[idx].y =  unewz_d[idx].x*nk[2];
  str_v3_d[idx].y =  unewy_d[idx].x*nk[2] + unewz_d[idx].x*nk[1];
  str_v4_d[idx].y =  unewx_d[idx].x*nk[2] + unewz_d[idx].x*nk[0];
  str_v5_d[idx].y =  unewx_d[idx].x*nk[1] + unewy_d[idx].x*nk[0];
   
}

__global__ void Compute_eigsts0(hipDoubleComplex *dfdphi_d, double *eigsts,
                               double *Chom11_d, double *Chom12_d, 
                               double *Chom44_d, double *Chet11_d, 
                               double *Chet12_d, double *Chet44_d, 
                               int *ny_d, int *nz_d)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double hphi, e_temp, eig11, eig22, eig33, Ct11, Ct12, Ct44;

  e_temp = dfdphi_d[idx].x;

  hphi  = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  eig11 = (*epszero_d)*hphi;
  eig22 = (*epszero_d)*hphi;
  eig33 = (*epszero_d)*hphi;

  Ct11  =  (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
  Ct12  =  (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
  Ct44  =  (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));

  eigsts[idx] = Ct11*eig11 + 
                Ct12*eig22 +
                Ct12*eig33 ;     
}
__global__ void Compute_eigsts1(hipDoubleComplex *dfdphi_d, double *eigsts,
                               double *Chom11_d, double *Chom12_d, 
                               double *Chom44_d, double *Chet11_d, 
                               double *Chet12_d, double *Chet44_d, 
                               int *ny_d, int *nz_d)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double hphi, e_temp, eig11, eig22, eig33, Ct11, Ct12, Ct44;

  e_temp = dfdphi_d[idx].x;

  hphi  = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  eig11 = (*epszero_d)*hphi;
  eig22 = (*epszero_d)*hphi;
  eig33 = (*epszero_d)*hphi;

  Ct11  =  (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
  Ct12  =  (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
  Ct44  =  (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));

  eigsts[idx] = Ct12*eig11 + 
                Ct11*eig22 +
                Ct12*eig33 ;     
}
__global__ void Compute_eigsts2(hipDoubleComplex *dfdphi_d, double *eigsts,
                               double *Chom11_d, double *Chom12_d, 
                               double *Chom44_d, double *Chet11_d, 
                               double *Chet12_d, double *Chet44_d, 
                               int *ny_d, int *nz_d)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double hphi, e_temp, eig11, eig22, eig33, Ct11, Ct12, Ct44;

  e_temp = dfdphi_d[idx].x;

  hphi  = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  eig11 = (*epszero_d)*hphi;
  eig22 = (*epszero_d)*hphi;
  eig33 = (*epszero_d)*hphi;

  Ct11  =  (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
  Ct12  =  (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
  Ct44  =  (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));

  eigsts[idx] = Ct12*eig11 + 
                Ct12*eig22 +
                Ct11*eig33 ;     
}


__global__ void Compute_persts(double *Chom11_d, double *Chom12_d, 
                     double *Chom44_d, double *Chet11_d, double *Chet12_d,
                     double *Chet44_d, hipDoubleComplex *dfdphi_d, 
                     hipDoubleComplex *str_v0_d, hipDoubleComplex *str_v1_d, 
                     hipDoubleComplex *str_v2_d, hipDoubleComplex *str_v3_d, 
                     hipDoubleComplex *str_v4_d, hipDoubleComplex *str_v5_d, 
                     double *persts0, double *persts1, double *persts2, 
                     double *persts3, double *persts4, double *persts5, 
                     int *ny_d, int *nz_d)
{
   	 
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double e_temp, hphi;
  double str_v[6], Ct11, Ct12, Ct44;

  e_temp = dfdphi_d[idx].x;

  hphi  = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  Ct11  =  (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
  Ct12  =  (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
  Ct44  =  (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));

  str_v[0] = str_v0_d[idx].x;
  str_v[1] = str_v1_d[idx].x;
  str_v[2] = str_v2_d[idx].x;
  str_v[3] = str_v3_d[idx].x;
  str_v[4] = str_v4_d[idx].x;
  str_v[5] = str_v5_d[idx].x;

  persts0[idx] = Ct11*str_v[0] + 
                 Ct12*str_v[1] +
                 Ct12*str_v[2];     

  persts1[idx] = Ct12*str_v[0] + 
                 Ct11*str_v[1] +
                 Ct12*str_v[2];     

  persts2[idx] = Ct12*str_v[0] + 
                 Ct12*str_v[1] +
                 Ct11*str_v[2];     

  persts3[idx] = Ct44*str_v[3]; 
  persts4[idx] = Ct44*str_v[4]; 
  persts5[idx] = Ct44*str_v[5]; 
}

__global__ void Compute_homstr(double *hom_strain_v, double *S11_d, 
                               double *S12_d, double *S44_d, 
                               double *sigappl_v_d, double *avgeigsts0,
                               double *avgeigsts1, double *avgeigsts2,
                               double *avgpersts0, double *avgpersts1,
                               double *avgpersts2, double *avgpersts3,
                               double *avgpersts4, double *avgpersts5)
{
   hom_strain_v[0] = (*S11_d)*(sigappl_v_d[0] + *avgeigsts0 - *avgpersts0) + 
                     (*S12_d)*(sigappl_v_d[1] + *avgeigsts1 - *avgpersts1) +
                     (*S12_d)*(sigappl_v_d[2] + *avgeigsts2 - *avgpersts2);

   hom_strain_v[1] = (*S12_d)*(sigappl_v_d[0] + *avgeigsts0 - *avgpersts0) + 
                     (*S11_d)*(sigappl_v_d[1] + *avgeigsts1 - *avgpersts1) +
                     (*S12_d)*(sigappl_v_d[2] + *avgeigsts2 - *avgpersts2);

   hom_strain_v[2] = (*S12_d)*(sigappl_v_d[0] + *avgeigsts0 - *avgpersts0) + 
                     (*S12_d)*(sigappl_v_d[1] + *avgeigsts1 - *avgpersts1) +
                     (*S11_d)*(sigappl_v_d[2] + *avgeigsts2 - *avgpersts2);

   hom_strain_v[3] = (*S44_d)*(sigappl_v_d[3] - *avgpersts3); 

   hom_strain_v[4] = (*S44_d)*(sigappl_v_d[4] - *avgpersts4); 

   hom_strain_v[5] = (*S44_d)*(sigappl_v_d[5] - *avgpersts5); 
}

__global__ void Compute_ts(hipDoubleComplex *ts0_d, hipDoubleComplex *ts1_d,
                           hipDoubleComplex *ts2_d, hipDoubleComplex *ts3_d,
                           hipDoubleComplex *ts4_d, hipDoubleComplex *ts5_d,
                       hipDoubleComplex *str_v0_d, hipDoubleComplex *str_v1_d,
                       hipDoubleComplex *str_v2_d, hipDoubleComplex *str_v3_d,
                       hipDoubleComplex *str_v4_d, hipDoubleComplex *str_v5_d,
                       double *epszero_d, double *Chom11_d, double *Chom12_d,
                       double *Chom44_d, double *Chet11_d, double *Chet12_d, 
                       double *Chet44_d, double *hom_strain_v , int *ny_d, 
                       int *nz_d)
{
   	
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double hphi, etemp, Ct11, Ct12, Ct44, Cin11, Cin12, C44in;
  double temp_v[6], eig11, eig22, eig33;

  hphi  = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);

  Ct11  = (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
  Ct12  = (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
  Ct44  = (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));
   
  Cin11 = (*Chet11_d*(2.0*hphi - 1.0));
  Cin12 = (*Chet12_d*(2.0*hphi - 1.0));
  Cin44 = (*Chet44_d*(2.0*hphi - 1.0));

  eig11 = (*epszero_d)*hphi;
  eig22 = (*epszero_d)*hphi;
  eig33 = (*epszero_d)*hphi;

  temp_v[0] = str_v0_d[idx].x;
  temp_v[1] = str_v1_d[idx].x;
  temp_v[2] = str_v2_d[idx].x;
  temp_v[3] = str_v3_d[idx].x;
  temp_v[4] = str_v4_d[idx].x;
  temp_v[5] = str_v5_d[idx].x;

  ts0_d[idx].x = Ct11*(eig11-(hom_strain_v[0]))-
                 Cin11*temp_v[0]+
                 Ct12*(eig22-(hom_strain_v[1]))-
                 Cin12*temp_v[1]+
                 Ct12*(eig33-(hom_strain_v[2]))-
                 Cin12*temp_v[2];

  ts1_d[idx].x = Ct12*(eig11-(hom_strain_v[0]))-
                 Cin12*temp_v[0]+
                 Ct11*(eig22-(hom_strain_v[1]))-
                 Cin11*temp_v[1]+
                 Ct12*(eig33-(hom_strain_v[2]))-
                 Cin12*temp_v[2];

  ts2_d[idx].x = Ct12*(eig11-(hom_strain_v[0]))-
                 Cin12*temp_v[0]+
                 Ct12*(eig22-(hom_strain_v[1]))-
                 Cin12*temp_v[1]+
                 Ct11*(eig33-(hom_strain_v[2]))-
                 Cin11*temp_v[2];

  ts3_d[idx].x = Ct44*(-1.0*(hom_strain_v[3]))-
                 Cin44*temp_v[3];

  ts4_d[idx].x = Ct44*(-1.0*(hom_strain_v[4]))-
                 Cin44*temp_v[4];

  ts5_d[idx].x = Ct44*(-1.0*(hom_strain_v[5]))-
                 Cin44*temp_v[5];
            
  ts0_d[idx].y = 0.0;
  ts1_d[idx].y = 0.0;
  ts2_d[idx].y = 0.0;
  ts3_d[idx].y = 0.0;
  ts4_d[idx].y = 0.0;
  ts5_d[idx].y = 0.0;
}

__global__ void Update_disp(int *ny_d, int *nz_d, 
                            double *kx_d, double *ky_d, double *kz_d, 
                            double *omega_v0, double *omega_v1,
                            double *omega_v2, double *omega_v3, 
                            double *omega_v4, double *omega_v5, 
                            hipDoubleComplex *ts0_d, hipDoubleComplex *ts1_d,
                            hipDoubleComplex *ts2_d, hipDoubleComplex *ts3_d, 
                            hipDoubleComplex *ts4_d, hipDoubleComplex *ts5_d,
                            hipDoubleComplex *unewx_d, 
                            hipDoubleComplex *unewy_d, 
                            hipDoubleComplex *unewz_d)
{
   	
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double          nk[3], omega[6];
  hipDoubleComplex stmp_v[6], fk10, fk20, fk30;

  nk[0] = kx_d[i];
  nk[1] = ky_d[j];
  nk[2] = kz_d[k];

  omega[0] = omega_v0[idx];
  omega[1] = omega_v1[idx];
  omega[2] = omega_v2[idx];
  omega[3] = omega_v3[idx];
  omega[4] = omega_v4[idx];
  omega[5] = omega_v5[idx];

  stmp_v[0].x = ts0_d[idx].x;    
  stmp_v[1].x = ts1_d[idx].x;    
  stmp_v[2].x = ts2_d[idx].x;    
  stmp_v[3].x = ts3_d[idx].x;    
  stmp_v[4].x = ts4_d[idx].x;    
  stmp_v[5].x = ts5_d[idx].x;

  stmp_v[0].y = ts0_d[idx].y;    
  stmp_v[1].y = ts1_d[idx].y;    
  stmp_v[2].y = ts2_d[idx].y;    
  stmp_v[3].y = ts3_d[idx].y;    
  stmp_v[4].y = ts4_d[idx].y;    
  stmp_v[5].y = ts5_d[idx].y;


  fk10.x = stmp_v[0].x * nk[0] + stmp_v[5].x * nk[1] + 
           stmp_v[4].x * nk[2];
  fk20.x = stmp_v[5].x * nk[0] + stmp_v[1].x * nk[1] + 
           stmp_v[3].x * nk[2];
  fk30.x = stmp_v[4].x * nk[0] + stmp_v[3].x * nk[1] + 
           stmp_v[2].x * nk[2];    
           
  fk10.y = stmp_v[0].y * nk[0] + stmp_v[5].y * nk[1] + 
           stmp_v[4].y * nk[2];
  fk20.y = stmp_v[5].y * nk[0] + stmp_v[1].y * nk[1] + 
           stmp_v[3].y * nk[2];
  fk30.y = stmp_v[4].y * nk[0] + stmp_v[3].y * nk[1] + 
           stmp_v[2].y * nk[2];

  unewx_d[idx].x = (omega[0] * fk10.y + 
                    omega[5] * fk20.y +
                    omega[4] * fk30.y);

  unewy_d[idx].x = (omega[5] * fk10.y + 
                    omega[1] * fk20.y +
                    omega[3] * fk30.y);
 
  unewz_d[idx].x = (omega[4] * fk10.y + 
                    omega[3] * fk20.y +
                    omega[2] * fk30.y);

  unewx_d[idx].y = -1.0*(omega[0] * fk10.x + 
                         omega[5] * fk20.x +
                         omega[4] * fk30.x);

  unewy_d[idx].y = -1.0*(omega[5] * fk10.x + 
                         omega[1] * fk20.x +
                         omega[3] * fk30.x);

  unewz_d[idx].y = -1.0*(omega[4] * fk10.x + 
                         omega[3] * fk20.x +
                         omega[2] * fk30.x);
}

__global__ void Compute_sq_diff_disp(hipDoubleComplex *unewx_d, 
                                     hipDoubleComplex *unewy_d, 
                                     hipDoubleComplex *unewz_d, 
                                     hipDoubleComplex *ux_d, 
                                     hipDoubleComplex *uy_d,
                                     hipDoubleComplex *uz_d,
                                     double *sq_diff_disp,
                                     int *ny_d, int *nz_d)
{

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  sq_diff_disp[idx] = pow((unewx_d[idx].x - ux_d[idx].x),2.0) +
                      pow((unewy_d[idx].x - uy_d[idx].x),2.0) +
                      pow((unewz_d[idx].x - uz_d[idx].x),2.0);

} 

__global__ void Copy_new_sol(hipDoubleComplex *unewx_d,hipDoubleComplex *unewy_d, 
                             hipDoubleComplex *unewz_d,hipDoubleComplex *ux_d, 
                             hipDoubleComplex *uy_d, hipDoubleComplex *uz_d,
                             int *ny_d, int *nz_d)
{

  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int k = threadIdx.z + blockDim.z*blockIdx.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));
   
  ux_d[idx].x = unewx_d[idx].x;
  uy_d[idx].x = unewy_d[idx].x;
  uz_d[idx].x = unewz_d[idx].x;

  ux_d[idx].y = unewx_d[idx].y;
  uy_d[idx].y = unewy_d[idx].y;
  uz_d[idx].y = unewz_d[idx].y;
}

__global__ void Compute_dfeldphi(hipDoubleComplex *str_v0_d, 
                         hipDoubleComplex *str_v1_d, hipDoubleComplex *str_v2_d,
                         hipDoubleComplex *str_v3_d, hipDoubleComplex *str_v4_d,
                         hipDoubleComplex *str_v5_d, hipDoubleComplex *dfdphi_d,
                         hipDoubleComplex *dfeldphi_d, double *Chom11_d, 
                         double *Chom12_d, double *Chom44_d, double *Chet11_d,
                         double *Chet12_d, double *Chet44_d, double *epszero_d, 
                         double *hom_strain_v, int *ny_d, int *nz_d)
{

   int i = threadIdx.x + blockDim.x*blockIdx.x;
   int j = threadIdx.y + blockDim.y*blockIdx.y;
   int k = threadIdx.z + blockDim.z*blockIdx.z;

   int idx = k + (*nz_d)*(j + i*(*ny_d));

   double hphi, hphi_p, e_temp, str_v[6], estr[3], hstr[6];
   double Ct11, Ct12, Ct44;

   e_temp   = dfdphi_d[idx].x;
   
   hphi     = e_temp*e_temp*e_temp*(6.0*e_temp*e_temp - 15.0*e_temp + 10.0);
   hphi_p   = (30.0*e_temp*e_temp*(1.0-e_temp)*(1.0-e_temp));           

   Ct11     = (*Chom11_d) + (*Chet11_d*(2.0*hphi - 1.0));
   Ct12     = (*Chom12_d) + (*Chet12_d*(2.0*hphi - 1.0));
   Ct44     = (*Chom44_d) + (*Chet44_d*(2.0*hphi - 1.0));
   
   str_v[0] = str_v0_d[idx].x;
   str_v[1] = str_v1_d[idx].x;
   str_v[2] = str_v2_d[idx].x;
   str_v[3] = str_v3_d[idx].x;
   str_v[4] = str_v4_d[idx].x;
   str_v[5] = str_v5_d[idx].x;

   estr[0]  = (*epszero_d)*hphi;
   estr[1]  = (*epszero_d)*hphi;
   estr[2]  = (*epszero_d)*hphi;
 
   hstr[0]  = hom_strain_v[0];
   hstr[1]  = hom_strain_v[1];
   hstr[2]  = hom_strain_v[2];
   hstr[3]  = hom_strain_v[3];
   hstr[4]  = hom_strain_v[4];
   hstr[5]  = hom_strain_v[5];


   dfeldphi_d[idx].x = 0.5*
               ((*Chet11_d)*2.0*hphi_p*
                   (hstr[0]+str_v[0]-estr[0])
                  *(hstr[0]+str_v[0]-estr[0])
               +(*Chet11_d)*2.0*hphi_p*
                   (hstr[1]+str_v[1]-estr[1])
                  *(hstr[1]+str_v[1]-estr[1])
               +(*Chet11_d)*2.0*hphi_p*
                   (hstr[2]+str_v[2]-estr[2])
                  *(hstr[2]+str_v[2]-estr[2])
               +2.0*(*Chet12_d)*2.0*hphi_p*
                   (hstr[0]+str_v[0]-estr[0])
                  *(hstr[1]+str_v[1]-estr[1])
               +2.0*(*Chet12_d)*2.0*hphi_p*
                   (hstr[0]+str_v[0]-estr[0])
                  *(hstr[2]+str_v[2]-estr[2])
               +2.0*(*Chet12_d)*2.0*hphi_p*
                   (hstr[1]+str_v[1]-estr[1])
                  *(hstr[2]+str_v[2]-estr[2])
               +(*Chet44_d)*2.0*hphi_p*
                   (hstr[3]+str_v[3])
                  *(hstr[3]+str_v[3])
               +(*Chet44_d)*2.0*hphi_p*
                   (hstr[4]+str_v[4])
                  *(hstr[4]+str_v[4])
               +(*Chet44_d)*2.0*hphi_p*
                   (hstr[5]+str_v[5])
                  *(hstr[5]+str_v[5])) -
               (
                Ct11*
                (hstr[0]+str_v[0]-estr[0])*
                (*epszero_d)*hphi_p +
                Ct11*
                (hstr[1]+str_v[1]-estr[1])*
                (*epszero_d)*hphi_p +
                Ct11*
                (hstr[2]+str_v[2]-estr[2])*
                (*epszero_d)*hphi_p +
                Ct12*
                (hstr[1]+str_v[1]-estr[1])*
                (*epszero_d)*hphi_p +
                Ct12*
                (hstr[1]+str_v[1]-estr[1])*
                (*epszero_d)*hphi_p +
                Ct12*
                (hstr[0]+str_v[0]-estr[0])*
                (*epszero_d)*hphi_p +
                Ct12*
                (hstr[0]+str_v[0]-estr[0])*
                (*epszero_d)*hphi_p +
                Ct12*
                (hstr[2]+str_v[2]-estr[2])*
                (*epszero_d)*hphi_p + 
                Ct12*
                (hstr[2]+str_v[2]-estr[2])*
                (*epszero_d)*hphi_p
               );

     dfeldphi_d[idx].y = 0.0; 
} 
__global__ void Average(double *x, double *sizescale)
{
   *x = *x*(*sizescale);
}
/*__global__ void FindTotalstr(hipDoubleComplex *str_v0_d, double *eigstr0,
                             double *hom_strain_v, double *totalstr_d,
                             int *ny_d, int *nz_d)
{
   
   int i = threadIdx.x + blockDim.x*blockIdx.x;
   int j = threadIdx.y + blockDim.y*blockIdx.y;
   int k = threadIdx.z + blockDim.z*blockIdx.z;

   int idx = k + (*nz_d)*(j + i*(*ny_d));

   totalstr_d[idx] = hom_strain_v[2] + str_v0_d[idx].x - eigstr0[idx];
}*/

void InhomElast (void){

  int              converge=1,iter=1, FALSE = 0;
  int              complex_size,double_size;
  double           *Cavg11,*Cavg12,*Cavg44;
  double           *avgeigsts0, *avgeigsts1, *avgeigsts2;
  double           *avgpersts0, *avgpersts1, *avgpersts2;
  double           *avgpersts3, *avgpersts4, *avgpersts5;
  double           *hom_strain_v;
  double           *disperr_d, *sq_diff_disp;
//  double           *totalstr_d, *totalstr;
  void             *t_storage = NULL;
  size_t           t_storage_bytes = 0;

  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, Ctotal11,
                         Cavg11, nx*ny*nz);

  complex_size = nx*ny*nz*sizeof(hipDoubleComplex);
  double_size  = nx*ny*nz*sizeof(double);

  //totalstr = (double*)malloc(double_size);

  checkCudaErrors(hipMalloc((void**)&str_v0_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&str_v1_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&str_v2_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&str_v3_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&str_v4_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&str_v5_d, complex_size));
  //checkCudaErrors(hipMalloc((void**)&totalstr_d, complex_size));

  checkCudaErrors(hipMalloc((void**)&unewx_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&unewy_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&unewz_d, complex_size));

  checkCudaErrors(hipMalloc((void**)&ts0_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&ts1_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&ts2_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&ts3_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&ts4_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&ts5_d, complex_size));

  //checkCudaErrors(hipMalloc((void**)&Cinhom11, double_size)); 
  //checkCudaErrors(hipMalloc((void**)&Cinhom12, double_size)); 
  //checkCudaErrors(hipMalloc((void**)&Cinhom44, double_size)); 

  checkCudaErrors(hipMalloc((void**)&Ctotal11, double_size)); 
  checkCudaErrors(hipMalloc((void**)&Ctotal12, double_size)); 
  checkCudaErrors(hipMalloc((void**)&Ctotal44, double_size)); 

  checkCudaErrors(hipMalloc((void**)&Cavg11, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&Cavg12, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&Cavg44, sizeof(double))); 

  checkCudaErrors(hipMalloc((void**)&avgeigsts0, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgeigsts1, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgeigsts2, sizeof(double))); 

  checkCudaErrors(hipMalloc((void**)&avgpersts0, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgpersts1, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgpersts2, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgpersts3, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgpersts4, sizeof(double))); 
  checkCudaErrors(hipMalloc((void**)&avgpersts5, sizeof(double))); 

  //checkCudaErrors(hipMalloc((void**)&eigstr0, double_size));
  //checkCudaErrors(hipMalloc((void**)&eigstr1, double_size));
  //checkCudaErrors(hipMalloc((void**)&eigstr2, double_size));

  checkCudaErrors(hipMalloc((void**)&eigsts, double_size));

  checkCudaErrors(hipMalloc((void**)&persts0, double_size));
  checkCudaErrors(hipMalloc((void**)&persts1, double_size));
  checkCudaErrors(hipMalloc((void**)&persts2, double_size));
  checkCudaErrors(hipMalloc((void**)&persts3, double_size));
  checkCudaErrors(hipMalloc((void**)&persts4, double_size));
  checkCudaErrors(hipMalloc((void**)&persts5, double_size));
  checkCudaErrors(hipMalloc((void**)&sq_diff_disp, double_size));
  checkCudaErrors(hipMalloc((void**)&disperr_d, sizeof(double)));

  checkCudaErrors(hipMalloc((void**)&hom_strain_v, 6*sizeof(double)));
  checkCudaErrors(hipMalloc(&t_storage, t_storage_bytes));

 /*----------------------------------------------------------------------
  *     Defining total elastic constants in Voight's form
  *----------------------------------------------------------------------*/

  Compute_Ctotal<<< Gridsize, Blocksize >>>(dfdphi_d, ny_d, nz_d, 
                                            Ctotal11, Ctotal12, Ctotal44, 
                                            Chom11_d, Chom12_d, Chom44_d, 
                                            Chet11_d, Chet12_d, Chet44_d);

 /*-----------------------------------------------------------------------
  *    Defining average elastic constants tensor in Voight's form
  *---------------------------------------------------------------------*/
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, Ctotal11,
                         Cavg11, nx*ny*nz);
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, Ctotal12, 
                 	 Cavg12, nx*ny*nz);
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, Ctotal44, 
			 Cavg44, nx*ny*nz);

  Average<<<1,1>>>(Cavg11, sizescale_d); 
  Average<<<1,1>>>(Cavg12, sizescale_d); 
  Average<<<1,1>>>(Cavg44, sizescale_d); 

  hipFree(Ctotal11);
  hipFree(Ctotal12);
  hipFree(Ctotal44);
 /*----------------------------------------------------------------------
  *                        Compliance tensor calculations 
  *---------------------------------------------------------------------*/
  Compute_Sij<<<1,1>>>(Cavg11, Cavg12, Cavg44, S11_d, S12_d, S44_d);

 /*Inhomogenous part of total stiffness tensor in Voight's form*/ 

  //Compute_Cinhom<<<Gridsize, Blocksize>>>(Cinhom11, Cinhom12, Cinhom44, 
  //                                        Ctotal11, Ctotal12, Ctotal44,
  //                                        Chom11_d, Chom12_d, Chom44_d, 
  //                                        ny_d, nz_d);
  //Finding eigen strain
  //Compute_eigstr<<<Gridsize, Blocksize>>>(dfdphi_d, eigstr0, eigstr1, 
  //                                        eigstr2, epszero_d, ny_d, nz_d);
  //Finding eigen stress
  Compute_eigsts0<<<Gridsize, Blocksize>>>(dfdphi_d, eigsts, Chom11_d, 
                                           Chom12_d, Chom44_d, Chet11_d, 
                                           Chet12_d, Chet44_d, ny_d, nz_d);
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, eigsts, 
			 avgeigsts0, nx*ny*nz);
  Compute_eigsts1<<<Gridsize, Blocksize>>>(dfdphi_d, eigsts, Chom11_d, 
                                           Chom12_d, Chom44_d, Chet11_d, 
                                           Chet12_d, Chet44_d, ny_d, nz_d);
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, eigsts, 
			 avgeigsts1, nx*ny*nz);
  Compute_eigsts2<<<Gridsize, Blocksize>>>(dfdphi_d, eigsts, Chom11_d, 
                                           Chom12_d, Chom44_d, Chet11_d, 
                                           Chet12_d, Chet44_d, ny_d, nz_d);
  hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, eigsts, 
			 avgeigsts2, nx*ny*nz);

  //Compute_eigsts<<<Gridsize, Blocksize>>>(Ctotal11, Ctotal12, Ctotal44, 
  //                                        eigsts0, eigsts1, eigsts2, 
  //                                        eigstr0, eigstr1, eigstr2,
  //                                        ny_d, nz_d);

  //Finding average eigen stress

  Average<<<1,1>>>(avgeigsts0, sizescale_d); 
  Average<<<1,1>>>(avgeigsts1, sizescale_d); 
  Average<<<1,1>>>(avgeigsts2, sizescale_d); 

  hipFree(eigsts);

  hipfftExecZ2Z(plan, ux_d, ux_d, HIPFFT_FORWARD);
  hipfftExecZ2Z(plan, uy_d, uy_d, HIPFFT_FORWARD);
  hipfftExecZ2Z(plan, uz_d, uz_d, HIPFFT_FORWARD);

  Compute_perstr<<< Gridsize, Blocksize >>>(ny_d, nz_d, kx_d, ky_d, kz_d, 
                                            ux_d, uy_d, uz_d, 
                                            str_v0_d, str_v1_d, str_v2_d,
                                            str_v3_d, str_v4_d, str_v5_d);

  hipfftExecZ2Z(plan, str_v0_d, str_v0_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v1_d, str_v1_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v2_d, str_v2_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v3_d, str_v3_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v4_d, str_v4_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v5_d, str_v5_d, HIPFFT_BACKWARD);

  hipfftExecZ2Z(plan, ux_d, ux_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, uy_d, uy_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, uz_d, uz_d, HIPFFT_BACKWARD);

  Normalize<<< Gridsize, Blocksize >>>(str_v0_d, sizescale_d, ny_d, nz_d); 
  Normalize<<< Gridsize, Blocksize >>>(str_v1_d, sizescale_d, ny_d, nz_d);  
  Normalize<<< Gridsize, Blocksize >>>(str_v2_d, sizescale_d, ny_d, nz_d);  
  Normalize<<< Gridsize, Blocksize >>>(str_v3_d, sizescale_d, ny_d, nz_d);  
  Normalize<<< Gridsize, Blocksize >>>(str_v4_d, sizescale_d, ny_d, nz_d);  
  Normalize<<< Gridsize, Blocksize >>>(str_v5_d, sizescale_d, ny_d, nz_d);

  Normalize<<< Gridsize, Blocksize >>>(ux_d, sizescale_d, ny_d, nz_d);
  Normalize<<< Gridsize, Blocksize >>>(uy_d, sizescale_d, ny_d, nz_d); 
  Normalize<<< Gridsize, Blocksize >>>(uz_d, sizescale_d, ny_d, nz_d); 


 /*-----------------------------------------------------------------------
  *                        Refinement of displacement
  *---------------------------------------------------------------------*/
  while (converge != FALSE){

      //Finidng periodic stresses
      Compute_persts<<< Gridsize, Blocksize>>> (Chom11_d, Chom12_d, Chom44_d, 
                                                Chet11_d, Chet12_d, Chet44_d, 
                                                dfdphi_d, str_v0_d, str_v1_d, 
                                                str_v2_d, str_v3_d, str_v4_d, 
                                                str_v5_d, persts0, persts1, 
                                                persts2, persts3, persts4, 
                                                persts5, ny_d, nz_d);

      //Finding average periodic stresses
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts0, 
			 avgpersts0, nx*ny*nz);
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts1, 
			 avgpersts1, nx*ny*nz);
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts2, 
			 avgpersts2, nx*ny*nz);
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts3, 
			 avgpersts3, nx*ny*nz);
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts4, 
			 avgpersts4, nx*ny*nz);
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, persts5, 
			 avgpersts5, nx*ny*nz);

      Average<<<1,1>>>(avgpersts0, sizescale_d);
      Average<<<1,1>>>(avgpersts1, sizescale_d);
      Average<<<1,1>>>(avgpersts2, sizescale_d);
      Average<<<1,1>>>(avgpersts3, sizescale_d);
      Average<<<1,1>>>(avgpersts4, sizescale_d);
      Average<<<1,1>>>(avgpersts5, sizescale_d);

      //Finding homogeneous strain
      Compute_homstr<<<1,1>>>(hom_strain_v, S11_d, S12_d, S44_d, sigappl_v_d,
                              avgeigsts0, avgeigsts1, avgeigsts2, 
                              avgpersts0, avgpersts1, avgpersts2, 
                              avgpersts3, avgpersts4, avgpersts5);

      //Finding ts
      Compute_ts<<< Gridsize, Blocksize>>>(ts0_d, ts1_d, ts2_d, 
                                           ts3_d, ts4_d, ts5_d, 
                                           str_v0_d, str_v1_d, str_v2_d, 
                                           str_v3_d, str_v4_d, str_v5_d, 
                                           epszero_d, Chom11_d, Chom12_d, 
                                           Chom44_d, Chet11_d, Chet12_d, 
                                           Chet44_d, hom_strain_v, ny_d, nz_d);
      /*Compute_ts<<<Gridsize, Blocksize>>>(Ctotal11, Ctotal12, Ctotal44, 
                                          Cinhom11, Cinhom12, Cinhom44, 
                                          ts0_d, ts1_d, ts2_d, 
                                          ts3_d, ts4_d, ts5_d,
                                          str_v0_d, str_v1_d, str_v2_d, 
                                          str_v3_d, str_v4_d, str_v5_d,
                                          eigstr0, eigstr1, eigstr2,
                                          hom_strain_v, ny_d, nz_d);
      */
      hipfftExecZ2Z(plan, ts0_d, ts0_d, HIPFFT_FORWARD);
      hipfftExecZ2Z(plan, ts1_d, ts1_d, HIPFFT_FORWARD);
      hipfftExecZ2Z(plan, ts2_d, ts2_d, HIPFFT_FORWARD);
      hipfftExecZ2Z(plan, ts3_d, ts3_d, HIPFFT_FORWARD);
      hipfftExecZ2Z(plan, ts4_d, ts4_d, HIPFFT_FORWARD);
      hipfftExecZ2Z(plan, ts5_d, ts5_d, HIPFFT_FORWARD);
   
      //Update displacements
      Update_disp<<< Gridsize,Blocksize >>>(ny_d, nz_d, kx_d, ky_d, kz_d, 
              omega_v0, omega_v1, omega_v2, omega_v3, omega_v4, omega_v5,
              ts0_d, ts1_d, ts2_d, ts3_d, ts4_d, ts5_d, 
              unewx_d, unewy_d, unewz_d);

      // Finding periodic strains
      Compute_perstr<<< Gridsize, Blocksize >>>(ny_d, nz_d, kx_d, ky_d, kz_d, 
              unewx_d, unewy_d, unewz_d, str_v0_d, str_v1_d, str_v2_d,
              str_v3_d, str_v4_d, str_v5_d);

      hipfftExecZ2Z(plan, unewx_d, unewx_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, unewy_d, unewy_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, unewz_d, unewz_d, HIPFFT_BACKWARD);

      hipfftExecZ2Z(plan, str_v0_d, str_v0_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, str_v1_d, str_v1_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, str_v2_d, str_v2_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, str_v3_d, str_v3_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, str_v4_d, str_v4_d, HIPFFT_BACKWARD);
      hipfftExecZ2Z(plan, str_v5_d, str_v5_d, HIPFFT_BACKWARD);
      
      Normalize<<< Gridsize, Blocksize >>>(unewx_d, sizescale_d, ny_d, nz_d);
      Normalize<<< Gridsize, Blocksize >>>(unewy_d, sizescale_d, ny_d, nz_d);
      Normalize<<< Gridsize, Blocksize >>>(unewz_d, sizescale_d, ny_d, nz_d);

      Normalize<<< Gridsize, Blocksize >>>(str_v0_d, sizescale_d, ny_d, nz_d); 
      Normalize<<< Gridsize, Blocksize >>>(str_v1_d, sizescale_d, ny_d, nz_d);  
      Normalize<<< Gridsize, Blocksize >>>(str_v2_d, sizescale_d, ny_d, nz_d);  
      Normalize<<< Gridsize, Blocksize >>>(str_v3_d, sizescale_d, ny_d, nz_d);  
      Normalize<<< Gridsize, Blocksize >>>(str_v4_d, sizescale_d, ny_d, nz_d);  
      Normalize<<< Gridsize, Blocksize >>>(str_v5_d, sizescale_d, ny_d, nz_d);

      //Find change in new and previous solution and save it to
      //unewx_d, unewy_d and unewz_d
      Compute_sq_diff_disp<<<Gridsize,Blocksize>>>(unewx_d, unewy_d, unewz_d, 
                                                   ux_d, uy_d, uz_d, 
                                                   sq_diff_disp, ny_d, 
                                                   nz_d);
  
      hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, sq_diff_disp, 
			 disperr_d, nx*ny*nz);
/*      double err_x, err_y, err_z;

      stat = hipblasDznrm2(blas_handle, nx*ny*nz, unewx_d, 1, &err_x); 
      stat = hipblasDznrm2(blas_handle, nx*ny*nz, unewy_d, 1, &err_y); 
      stat = hipblasDznrm2(blas_handle, nx*ny*nz, unewz_d, 1, &err_z);

      if (stat != HIPBLAS_STATUS_SUCCESS){
         
         printf("Error in finding norm");
 
      }

      disperror = sqrt(err_x*err_x + err_y*err_y + err_z*err_z); 
*/
      hipMemcpy(&disperror, disperr_d, sizeof(double), hipMemcpyDeviceToHost);
      disperror = sqrt(disperror); 
      //printf ("\niter=%d error = %le", iter, disperror);

      if (disperror < TOLERENCE){
        //printf("\nConvergence achieved at %d\n", iter);
        converge = 0;
      }
      
      iter = iter + 1;

      Copy_new_sol<<< Gridsize, Blocksize>>> (unewx_d, unewy_d, unewz_d,
                                              ux_d, uy_d, uz_d,
                                              ny_d, nz_d);
  }

/*
  hipfftExecZ2Z(plan, str_v0_d, str_v0_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v1_d, str_v1_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v2_d, str_v2_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v3_d, str_v3_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v4_d, str_v4_d, HIPFFT_BACKWARD);
  hipfftExecZ2Z(plan, str_v5_d, str_v5_d, HIPFFT_BACKWARD);
   
  
  Normalize<<< Gridsize, Blocksize >>>(str_v0_d, sizescale_d, ny_d, nz_d);      
  Normalize<<< Gridsize, Blocksize >>>(str_v1_d, sizescale_d, ny_d, nz_d);      
  Normalize<<< Gridsize, Blocksize >>>(str_v2_d, sizescale_d, ny_d, nz_d);      
  Normalize<<< Gridsize, Blocksize >>>(str_v3_d, sizescale_d, ny_d, nz_d);      
  Normalize<<< Gridsize, Blocksize >>>(str_v4_d, sizescale_d, ny_d, nz_d);      
  Normalize<<< Gridsize, Blocksize >>>(str_v5_d, sizescale_d, ny_d, nz_d);      
*/
  Compute_persts<<< Gridsize, Blocksize>>> (Chom11_d, Chom12_d, Chom44_d, 
                                            Chet11_d, Chet12_d, Chet44_d, 
                                            dfdphi_d, str_v0_d, str_v1_d, 
                                            str_v2_d, str_v3_d, str_v4_d, 
                                            str_v5_d, persts0, persts1, 
                                            persts2, persts3, persts4, 
                                            persts5, ny_d, nz_d);

 Compute_homstr<<<1,1>>>(hom_strain_v, S11_d, S12_d, S44_d, sigappl_v_d,
                          avgeigsts0, avgeigsts1, avgeigsts2, 
                          avgpersts0, avgpersts1, avgpersts2, 
                          avgpersts3, avgpersts4, avgpersts5);

 Compute_dfeldphi<<< Gridsize, Blocksize>>>(str_v0_d, str_v1_d, str_v2_d,
                                            str_v3_d, str_v4_d, str_v5_d, 
                                            dfdphi_d, dfeldphi_d, Chom11_d, 
                                            Chom12_d, Chom44_d, Chet11_d,
                                            Chet12_d, Chet44_d, epszero_d, 
                                            hom_strain_v, ny_d, nz_d);

  //FindTotalstr<<< Gridsize, Blocksize>>> (str_v2_d, eigstr2, hom_strain_v, totalstr_d, ny_d, nz_d);

  //hipMemcpy(totalstr, totalstr_d, double_size, hipMemcpyDeviceToHost);

  /*fp = fopen("strfield.txt","w");
  for (int k =0; k<nz; k++){
	
    fprintf(fp,"%d\t%le\n",k,totalstr[k+nz*(ny/2+ny*nx/2)]);
    
  }
     
  fclose(fp);*/

  hipFree(str_v0_d);
  hipFree(str_v1_d);
  hipFree(str_v2_d);
  hipFree(str_v3_d);
  hipFree(str_v4_d);
  hipFree(str_v5_d);
  hipFree(unewx_d);
  hipFree(unewy_d);
  hipFree(unewz_d);
  hipFree(ts0_d);
  hipFree(ts1_d);
  hipFree(ts2_d);
  hipFree(ts3_d);
  hipFree(ts4_d);
  hipFree(ts5_d);
  hipFree(avgeigsts0);
  hipFree(avgeigsts1);
  hipFree(avgeigsts2);
  hipFree(persts0);
  hipFree(persts1);
  hipFree(persts2);
  hipFree(persts3);
  hipFree(persts4);
  hipFree(persts5);
  hipFree(avgpersts0);
  hipFree(avgpersts1);
  hipFree(avgpersts2);
  hipFree(avgpersts3);
  hipFree(avgpersts4);
  hipFree(avgpersts5);
  hipFree(Cinhom11);
  hipFree(Cinhom12);
  hipFree(Cinhom44);
  hipFree(hom_strain_v);
  hipFree(Cavg11);
  hipFree(Cavg12);
  hipFree(Cavg44);
  hipFree(sq_diff_disp);
  hipFree(disperr_d);
  hipFree(t_storage);
  //hipFree(totalstr_d);
  //free(totalstr);
}
