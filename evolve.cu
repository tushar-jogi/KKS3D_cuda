#include "hip/hip_runtime.h"

__global__ void ComputeGreentensor(double *kx_d, double *ky_d, double *kz_d, 
                          double *Chom11_d, double *Chom12_d, double *Chom44_d,
                          int *nx_d, int *ny_d, int *nz_d, double *omega_v0,
                          double *omega_v1, double *omega_v2, double *omega_v3,
                          double *omega_v4, double *omega_v5)
{

  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k +(*nz_d)*(j + i*(*ny_d));

  double  adjomega[6], det_omega, invomega_v[6], n[3];
      
  n[0] = kx_d[i];
  n[1] = ky_d[j];
  n[2] = kz_d[k];

  invomega_v[0] = (*Chom11_d)*n[0]*n[0] + (*Chom44_d)*n[1]*n[1] +
                  (*Chom44_d)*n[2]*n[2];
  invomega_v[1] = (*Chom44_d)*n[0]*n[0] + (*Chom11_d)*n[1]*n[1] +
                  (*Chom44_d)*n[2]*n[2];
  invomega_v[2] = (*Chom44_d)*n[0]*n[0] + (*Chom44_d)*n[1]*n[1] +
                  (*Chom11_d)*n[2]*n[2];
  invomega_v[3] = ((*Chom12_d) + (*Chom44_d))*n[1]*n[2];
  invomega_v[4] = ((*Chom12_d) + (*Chom44_d))*n[0]*n[2];
  invomega_v[5] = ((*Chom12_d) + (*Chom44_d))*n[0]*n[1];

  det_omega = invomega_v[0]*(invomega_v[1]*invomega_v[2] - 
                             invomega_v[3]*invomega_v[3])-
              invomega_v[5]*(invomega_v[5]*invomega_v[2] -
                             invomega_v[4]*invomega_v[3])+
              invomega_v[4]*(invomega_v[5]*invomega_v[3] -
                             invomega_v[4]*invomega_v[1]);

  adjomega[0] = (invomega_v[1]*invomega_v[2]-
                 invomega_v[3]*invomega_v[3]);
  adjomega[1] = (invomega_v[0]*invomega_v[2]-
                 invomega_v[4]*invomega_v[4]);
  adjomega[2] = (invomega_v[0]*invomega_v[1]-
                 invomega_v[5]*invomega_v[5]);
  adjomega[3] =-(invomega_v[0]*invomega_v[3]-
                 invomega_v[4]*invomega_v[5]);
  adjomega[4] = (invomega_v[5]*invomega_v[3]-
                 invomega_v[4]*invomega_v[1]);
  adjomega[5] =-(invomega_v[5]*invomega_v[2]-
                 invomega_v[4]*invomega_v[3]);

  if (fabs(det_omega) > 1.0e-06){
     omega_v0[idx] = (1.0/det_omega)*adjomega[0];
     omega_v1[idx] = (1.0/det_omega)*adjomega[1];
     omega_v2[idx] = (1.0/det_omega)*adjomega[2];
     omega_v3[idx] = (1.0/det_omega)*adjomega[3];
     omega_v4[idx] = (1.0/det_omega)*adjomega[4];
     omega_v5[idx] = (1.0/det_omega)*adjomega[5];
  } 

  else{
     omega_v0[idx] = 0.0;
     omega_v1[idx] = 0.0;
     omega_v2[idx] = 0.0;
     omega_v3[idx] = 0.0;
     omega_v4[idx] = 0.0;
     omega_v5[idx] = 0.0;
  }

  __syncthreads();

}


__global__ void  ComputeGradphi(double *kx_d, double *ky_d, double *kz_d,
                                int *nx_d, int *ny_d, int *nz_d,
                                hipDoubleComplex *phi_d,
                                hipDoubleComplex *gradphix_d, 
                                hipDoubleComplex *gradphiy_d,
                                hipDoubleComplex *gradphiz_d)
{

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double  n[3];

  n[0] = kx_d[i];
  n[1] = ky_d[j];
  n[2] = kz_d[k];

  gradphix_d[idx].x = -1.0*n[0]*phi_d[idx].y;  
  gradphix_d[idx].y = n[0]*phi_d[idx].x;  
  gradphiy_d[idx].x = -1.0*n[1]*phi_d[idx].y;  
  gradphiy_d[idx].y = n[1]*phi_d[idx].x;  
  gradphiz_d[idx].x = -1.0*n[2]*phi_d[idx].y;  
  gradphiz_d[idx].y = n[2]*phi_d[idx].x;  

  __syncthreads();    

}


__global__ void ComputeDrivForce(hipDoubleComplex *comp_d, 
                                 hipDoubleComplex *dfdphi_d,
                                 hipDoubleComplex *gradphix_d, 
                                 hipDoubleComplex *gradphiy_d,
                                 hipDoubleComplex *gradphiz_d, 
                                 hipDoubleComplex *varmobx_d,
                                 hipDoubleComplex *varmoby_d, 
                                 hipDoubleComplex *varmobz_d,
                                 double *f0AVminv_d, double *f0BVminv_d, 
                                 double *c_beta_eq_d, double *c_alpha_eq_d, 
                                 double *diffusivity_d, double *w_d, int *ny_d,
                                 int *nz_d)
{

  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double  interp_phi, interp_prime, g_prime;
  double  ctemp, etemp;
  double  f_alpha, f_beta, mubar;
  double  A_by_B, B_by_A;
  double  calpha, cbeta;

  A_by_B = (*f0AVminv_d)/(*f0BVminv_d);
  B_by_A = (*f0BVminv_d)/(*f0AVminv_d);
   
  ctemp  = comp_d[idx].x;
  etemp  = dfdphi_d[idx].x;
   
  interp_phi   = etemp * etemp * etemp * 
                (6.0 * etemp * etemp - 15.0 * etemp + 10.0);
  interp_prime = 30.0 * etemp * etemp * pow((1.0 - etemp), 2.0);  
  g_prime      = 2.0 * etemp * (1.0 - etemp) * (1.0 - 2.0 * etemp);

  calpha       = (ctemp - interp_phi * 
                 ((*c_beta_eq_d) - (*c_alpha_eq_d) * A_by_B))/
                 (interp_phi*A_by_B + (1.0-interp_phi));

  cbeta        = (ctemp + (1.0 - interp_phi) * 
                 (B_by_A * (*c_beta_eq_d) - (*c_alpha_eq_d)))/
                 (interp_phi + B_by_A*(1.0 - interp_phi));

  comp_d[idx].x  = calpha*(1.0 - interp_phi) + 
                   cbeta*interp_phi; 
  comp_d[idx].y  = 0.0;

  f_alpha      = (*f0AVminv_d)*(calpha - (*c_alpha_eq_d))* 
                 (calpha - (*c_alpha_eq_d));
  f_beta       = (*f0BVminv_d)*(cbeta - (*c_beta_eq_d))* 
                 (cbeta - (*c_beta_eq_d));
   
  varmobx_d[idx].x = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphix_d[idx].x;
  varmobx_d[idx].y = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphix_d[idx].y;
  varmoby_d[idx].x = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphiy_d[idx].x;
  varmoby_d[idx].y = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphiy_d[idx].y;
  varmobz_d[idx].x = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphiz_d[idx].x;
  varmobz_d[idx].y = (*diffusivity_d)*interp_prime* 
                     (calpha - cbeta) * gradphiz_d[idx].y;
   
  mubar = 2.0 * (*f0BVminv_d) * (cbeta - (*c_beta_eq_d));

  dfdphi_d[idx].x = interp_prime*(f_beta - f_alpha + 
                    (calpha - cbeta)*mubar) + 
                    (*w_d)*g_prime; 
  dfdphi_d[idx].y = 0.0;

  __syncthreads();

}

__global__ void  ComputeDfdc(hipDoubleComplex *dfdc_d, 
                             hipDoubleComplex *varmobx_d, 
                             hipDoubleComplex *varmoby_d, 
                             hipDoubleComplex *varmobz_d,
                             int *nx_d, int *ny_d, int *nz_d, 
                             double *kx_d, double *ky_d, 
                             double *kz_d)
{

  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));
  double  n[3];

  n[0] = kx_d[i];
  n[1] = ky_d[j];
  n[2] = kz_d[k];

  dfdc_d[idx].x = -1.0*(n[0]*varmobx_d[idx].y + 
                        n[1]*varmoby_d[idx].y +
                        n[2]*varmobz_d[idx].y); 
 
  dfdc_d[idx].y = (n[0]*varmobx_d[idx].x + 
                   n[1]*varmoby_d[idx].x +
                   n[2]*varmobz_d[idx].x);
  
  __syncthreads();    

}

__global__ void Update_comp_phi (hipDoubleComplex *comp_d, 
                                 hipDoubleComplex *dfdc_d, 
                                 hipDoubleComplex *phi_d, 
                                 hipDoubleComplex *dfdphi_d,
                                 hipDoubleComplex *dfeldphi_d, double *kx_d, 
                                 double *ky_d, double *kz_d, double *dt_d,
                                 double *diffusivity_d, double *kappa_phi_d, 
                                 double *relax_coeff_d, int *elast_int_d, 
                                 int *nx_d, int *ny_d, int *nz_d)
{
  
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  double           kpow2, lhs, lhse;
  hipDoubleComplex  rhs, rhse; 
 
  kpow2 = kx_d[i]*kx_d[i] + ky_d[j]*ky_d[j] + kz_d[k]*kz_d[k];

  lhs = 1.0 + (*diffusivity_d)*kpow2*(*dt_d);   

  rhs.x = comp_d[idx].x + (*dt_d)*dfdc_d[idx].x;
  rhs.y = comp_d[idx].y + (*dt_d)*dfdc_d[idx].y;

  comp_d[idx].x = rhs.x/lhs; 
  comp_d[idx].y = rhs.y/lhs;
  
  lhse = 1.0 + 2.0*(*relax_coeff_d)*(*kappa_phi_d)*kpow2*(*dt_d);

  if ((*elast_int_d) == 1 ){

     rhse.x  = phi_d[idx].x - (*relax_coeff_d)*(*dt_d)*
               (dfdphi_d[idx].x + dfeldphi_d[idx].x);
     rhse.y  = phi_d[idx].y - (*relax_coeff_d)*(*dt_d)*
               (dfdphi_d[idx].y + dfeldphi_d[idx].y);

  }
  else{

     rhse.x  = phi_d[idx].x - (*relax_coeff_d)*(*dt_d)*
              (dfdphi_d[idx].x);
     rhse.y  = phi_d[idx].y - (*relax_coeff_d)*(*dt_d)*
              (dfdphi_d[idx].y);

  }

  phi_d[idx].x = rhse.x/lhse;
  phi_d[idx].y = rhse.y/lhse;

  dfdphi_d[idx].x = phi_d[idx].x;
  dfdphi_d[idx].y = phi_d[idx].y;
 

}

__global__ void Normalize(hipDoubleComplex *x, double *sizescale_d, int *ny_d,
                          int *nz_d)
{

  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));
 
  x[idx].x = x[idx].x * (*sizescale_d);
  x[idx].y = x[idx].y * (*sizescale_d);

}

__global__ void SaveReal(double *temp, hipDoubleComplex *x, int *ny_d,
                         int *nz_d)
{
  
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  temp[idx] = x[idx].x; 

}
__global__ void Find_err_matrix(double *temp, double *diff, 
                                hipDoubleComplex *comp_d, int *ny_d,
                                int *nz_d, double *c0_d)
{
  
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  int k = threadIdx.z + blockIdx.z*blockDim.z;

  int idx = k + (*nz_d)*(j + i*(*ny_d));

  diff[idx] = fabs(comp_d[idx].x - temp[idx]);

  //if (idx == 0){
  //   if (fabs(comp_d[idx].x - *c0_d)<1.0e-05)
  //     *exit_cond_d = 0;
  //   else
  //     *exit_cond_d = 1; 
  //}      
  
}    
void Evolve(void)
{
  void Output_Conf (int steps);
  void Calc_uzero(void);
  void InhomElast(void);
  //void HomElast(void);
  int       loop_condition, count;
  double    *temp_real;
  double    *kx, *ky, *kz;
  double    *tempreal_d, *temp_diff ;
  double    maxerror, *maxerr_d;
  double    f0AVminv, f0BVminv;
  double    *f0AVminv_d, *f0BVminv_d;
  hipfftDoubleComplex    comp_at_corner;
  int       *exit_cond_d;
  void      *t_storage = NULL;
  size_t    t_storage_bytes = 0;
  size_t    complex_size, double_size;

  hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, temp_diff, maxerr_d,
                         nx*ny*nz);

  complex_size = nx*ny*nz*sizeof(hipDoubleComplex);
  double_size  = nx*ny*nz*sizeof(double);

  f0AVminv = f0A * (1.0/Vm) ;
  f0BVminv = f0B * (1.0/Vm) ;

  checkCudaErrors(hipMalloc((void**)&tempreal_d, double_size));
  checkCudaErrors(hipMalloc((void**)&temp_diff,  double_size));
  checkCudaErrors(hipMalloc((void**)&maxerr_d, sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&exit_cond_d, sizeof(int)));
  checkCudaErrors(hipMalloc((void**)&f0AVminv_d, sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&f0BVminv_d, sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&S11_d, sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&S12_d, sizeof(double)));
  checkCudaErrors(hipMalloc((void**)&S44_d, sizeof(double)));
  checkCudaErrors(hipMalloc(&t_storage, t_storage_bytes));

  checkCudaErrors(hipMemcpy(f0AVminv_d, &f0AVminv, sizeof(double),
        hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(f0BVminv_d, &f0BVminv, sizeof(double),
        hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(comp, comp_d, complex_size,
        hipMemcpyDeviceToHost));

  SaveReal<<< Gridsize, Blocksize >>>(tempreal_d, comp_d, ny_d, nz_d);

 //Fourier vectors: defined in host and copied to device memory 
  kx = (double*) malloc(nx*sizeof(double));
  ky = (double*) malloc(ny*sizeof(double));
  kz = (double*) malloc(nz*sizeof(double));

  for (int i = 0 ; i < nx ; i++ ) {
    if (i < nx/2) 
      kx[i] = (double) i * dkx;
    else 
      kx[i] = (double)(i-nx) * dkx;
  } 

  for (int j = 0; j < ny; j++){
    if (j < ny/2)
      ky[j] = (double)j * dky;
    else
      ky[j] = (double)(j-ny) * dky;
  }

  for (int k = 0; k < nz; k++){
    if (k < nz/2)
      kz[k] = (double)k * dkz;
    else
      kz[k] = (double)(k-nz) * dkz;
  }

  checkCudaErrors(hipMemcpy(kx_d, kx, nx*sizeof(double),
        hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(ky_d, ky, ny*sizeof(double),
        hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(kz_d, kz, nz*sizeof(double),
        hipMemcpyHostToDevice));

  free(kx);
  free(ky);
  free(kz);

  checkCudaErrors(hipMalloc((void**)&gradphix_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&gradphiy_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&gradphiz_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&varmobx_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&varmoby_d, complex_size));
  checkCudaErrors(hipMalloc((void**)&varmobz_d, complex_size));

  if (elast_int == 1)
    ComputeGreentensor<<< Gridsize, Blocksize >>>(kx_d, ky_d, kz_d, Chom11_d, 
                                                  Chom12_d, Chom44_d,
                                                  nx_d, ny_d, nz_d, omega_v0,
                                                  omega_v1, omega_v2,
                                                  omega_v3, omega_v4,
                                                  omega_v5);

  if (inhom != 1){

    S11 = ((Chom11)+(Chom12))/((Chom11)*(Chom11) + (Chom11)*(Chom12) -
          2.0*(Chom12)*(Chom12));
    S12 = (-1.0*(Chom12))/((Chom11)*(Chom11) + (Chom11)*(Chom12) -
          2.0*(Chom12)*(Chom12));
    S44 = 1.0/(Chom44);

    checkCudaErrors(hipMemcpy(S11_d, &S11, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(S12_d, &S12, sizeof(double),
          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(S44_d, &S44, sizeof(double),
          hipMemcpyHostToDevice));

  }

  if (hipfftExecZ2Z(plan, phi_d, phi_d, HIPFFT_FORWARD)!= HIPFFT_SUCCESS)
    printf("fft failed");

  hipGetLastError();

  printf("%s\n",hipGetErrorString(Err));

  iteration = 1;
  loop_condition = 1;

  //Time loop
  for (count = initcount; count <= num_steps; count++) {
  
    if (((count % t_prof1)==0 && count <= numsteps_prof1) ||
        ((count % t_prof2)==0 && count  > numsteps_prof1) ||  
         (count == num_steps) || 
         (loop_condition == 0)) {

      printf ("total_time=%le\n", sim_time);
      printf ("writing configuration to file!\n");

      checkCudaErrors(hipMemcpy(comp, comp_d, complex_size,
            hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(dfdphi, dfdphi_d, complex_size,
            hipMemcpyDeviceToHost));

      Output_Conf(count);

    }
 
    if (count > num_steps || loop_condition == 0)
      break;
   
    printf("Iteration No: %d\n",iteration);

      //Finding elastic driving force in real space
    if (elast_int == 1 && count >= time_elast ){

      if (count == initcount)
        Calc_uzero();
      //if (inhom == 1)
        InhomElast();
      //else
       // HomElast();
    }


    ComputeGradphi<<< Gridsize, Blocksize >>>(kx_d, ky_d, kz_d, 
                                            nx_d, ny_d, nz_d, 
                                            phi_d, gradphix_d, gradphiy_d, 
                                            gradphiz_d);

    hipfftExecZ2Z(plan, gradphix_d, gradphix_d, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan, gradphiy_d, gradphiy_d, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan, gradphiz_d, gradphiz_d, HIPFFT_BACKWARD);
    
    Normalize<<< Gridsize, Blocksize >>>(gradphix_d, sizescale_d, ny_d, nz_d);
    Normalize<<< Gridsize, Blocksize >>>(gradphiy_d, sizescale_d, ny_d, nz_d);
    Normalize<<< Gridsize, Blocksize >>>(gradphiz_d, sizescale_d, ny_d, nz_d);

    ComputeDrivForce<<< Gridsize, Blocksize >>>(comp_d, dfdphi_d, 
                                gradphix_d, gradphiy_d, gradphiz_d, 
                                varmobx_d, varmoby_d, varmobz_d, 
                                f0AVminv_d, f0BVminv_d, c_beta_eq_d, 
                                c_alpha_eq_d, diffusivity_d, w_d, ny_d, nz_d);

    if (hipfftExecZ2Z(plan,varmobx_d, varmobx_d,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
       printf("fft failed\n");
    if (hipfftExecZ2Z(plan,varmoby_d, varmoby_d,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
       printf("fft failed\n");
    if (hipfftExecZ2Z(plan,varmobz_d, varmobz_d,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
       printf("fft failed\n");

    ComputeDfdc<<< Gridsize, Blocksize >>>(dfdc_d, varmobx_d, varmoby_d,
                                           varmobz_d, nx_d, ny_d, nz_d, 
                                           kx_d, ky_d, kz_d);


    hipfftExecZ2Z(plan, comp_d,     comp_d, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan, dfdphi_d, dfdphi_d, HIPFFT_FORWARD);

    if (elast_int == 1 && count > time_elast)
      hipfftExecZ2Z(plan, dfeldphi_d, dfeldphi_d, HIPFFT_FORWARD);
    
    Update_comp_phi<<< Gridsize, Blocksize >>>(comp_d,dfdc_d,phi_d,dfdphi_d,
                                 dfeldphi_d, kx_d, ky_d, kz_d, dt_d,
                                 diffusivity_d, kappa_phi_d, relax_coeff_d,
                                 elast_int_d, nx_d, ny_d, nz_d);

    hipfftExecZ2Z(plan, comp_d,     comp_d, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan, dfdphi_d, dfdphi_d, HIPFFT_BACKWARD);

    Normalize<<< Gridsize, Blocksize >>>(comp_d,   sizescale_d, ny_d, nz_d);
    Normalize<<< Gridsize, Blocksize >>>(dfdphi_d, sizescale_d, ny_d, nz_d);

    Find_err_matrix<<< Gridsize, Blocksize >>>(tempreal_d, temp_diff, comp_d, 
                       ny_d, nz_d, c0_d);

    hipMemcpy(&comp_at_corner,comp_d,sizeof(hipfftDoubleComplex),
                hipMemcpyDeviceToHost);
    printf("comp_at_corner = %lf\n", Re(comp_at_corner));
    if (fabs(Re(comp_at_corner) - c0) >= 1.0e-04)
    {
	printf("Growth condition has vanished!!!!\n");
        exit(0);
    }
    hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, temp_diff, maxerr_d, 
                           nx*ny*nz);       

    if (loop_condition == 0)
      printf("Simulation Converged");

    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(&maxerror, maxerr_d, sizeof(double),
          hipMemcpyDeviceToHost));

    if(maxerror <= Tolerance){
      printf("Microstructure converged\n");
      loop_condition = 0;      
    }
    sim_time = sim_time + dt;

    SaveReal <<< Gridsize, Blocksize >>> (tempreal_d, comp_d, ny_d, nz_d);

    iteration = iteration + 1;
  }//time loop ends

  hipFree(tempreal_d);
  hipFree(temp_diff);
  hipFree(maxerr_d);
  hipFree(exit_cond_d);
  hipFree(f0AVminv_d);
  hipFree(f0BVminv_d);
  hipFree(t_storage);
  hipFree(S11_d);
  hipFree(S12_d);
  hipFree(S44_d);
  hipFree(varmobx_d);
  hipFree(varmoby_d);
  hipFree(varmobz_d);
  hipFree(gradphix_d);
  hipFree(gradphiy_d);
  hipFree(gradphiz_d);
}

#include "out_conf.cu"
#include "calc_uzero.cu"
#include "inhomelast.cu"
